#include "Main.cuh"

int main()
{
	int a[] = { 1,2,3 };
	int b[] = { 1,2,3 };
	int c[sizeof(a) / sizeof(int)] = { 0 };

	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;

	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));

	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

	vectorAdd << < 1, sizeof(a) / sizeof(int) >> > (cudaA, cudaB, cudaC);

	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

	// Need triangle and ray
	Triangle h_triangle;
	h_triangle.vertecies[0] = { 0,0,0 };
	h_triangle.vertecies[1] = { 1,1,0 };
	h_triangle.vertecies[2] = { 2,0,0 };
	h_triangle.normal = { 0,0,-1 };


	Ray h_ray;
	h_ray.origin = { 0, 0, 1 };
	h_ray.direction = { 0,0,1 };

	Ray* d_ray;
	Triangle* d_triangle;

	hipMalloc(&d_ray, sizeof(Ray));
	hipMalloc(&d_triangle, sizeof(Triangle));

	hipMemcpy(d_ray, &h_ray, sizeof(Ray), hipMemcpyHostToDevice);
	hipMemcpy(d_triangle, &h_triangle, sizeof(Triangle), hipMemcpyHostToDevice);

	rayCastCUDA << <1, 1 >> > (d_ray, d_triangle);

	hipMemcpy(&h_ray, d_ray, sizeof(Ray), hipMemcpyDeviceToHost);

	std::cout << h_ray.collisionPoint.x;
	std::cout << h_ray.collisionPoint.y;
	std::cout << h_ray.collisionPoint.z;

	printf("%d\n", c[1]);

	return;

}