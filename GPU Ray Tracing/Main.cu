#include "Main.cuh"

int main()
{
	int a[] = { 1,2,3 };
	int b[] = { 1,2,3 };
	int c[sizeof(a) / sizeof(int)] = { 0 };

	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;

	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));

	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

	vectorAdd << < 1, sizeof(a) / sizeof(int) >> > (cudaA, cudaB, cudaC);

	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

	// Need triangle and ray
	Triangle h_triangle;
	h_triangle.vertecies[0] = { 0,0,0 };
	h_triangle.vertecies[1] = { 1,1,0 };
	h_triangle.vertecies[2] = { 2,0,0 };
	h_triangle.normal = h_triangle.vertecies[1].subtract(h_triangle.vertecies[0]).cross(h_triangle.vertecies[2].subtract(h_triangle.vertecies[0])).normalised();



	Ray h_ray;
	h_ray.origin = { 0, 0, -1 };
	h_ray.direction = { 0,0,1 };

	Ray* d_ray;
	Triangle* d_triangle;

	hipMalloc(&d_ray, sizeof(Ray));
	hipMalloc(&d_triangle, sizeof(Triangle));

	hipMemcpy(d_ray, &h_ray, sizeof(Ray), hipMemcpyHostToDevice);
	hipMemcpy(d_triangle, &h_triangle, sizeof(Triangle), hipMemcpyHostToDevice);

	rayCastCUDA << < 1, 1, >> > (h_ray, h_triangle);

	hipMemcpy(&h_ray, d_ray, sizeof(Ray), hipMemcpyDeviceToHost);

	printf("%d\n", c[1]);

	return;

}