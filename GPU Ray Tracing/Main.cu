#include "Main.cuh"

void renderScreen(Camera* camera)
{
	Ray* CUDA_rays = camera->rays_through_screen;
	Triangle* CUDA_triangles = camera->scene.triangles.data();
	int HOST_num_of_triangles = static_cast<int>(camera->scene.triangles.size());
	int* CUDA_num_of_triangles = &HOST_num_of_triangles;

	hipMalloc(&CUDA_rays, sizeof(CUDA_rays));
	hipMalloc(&CUDA_triangles, sizeof(CUDA_triangles));
	hipMalloc(&CUDA_num_of_triangles, sizeof(CUDA_num_of_triangles));

	hipMemcpy(CUDA_rays, camera->rays_through_screen, sizeof(CUDA_rays), hipMemcpyHostToDevice);
	hipMemcpy(CUDA_triangles, camera->scene.triangles.data(), sizeof(CUDA_triangles), hipMemcpyHostToDevice);

	traceRay <<< 1, camera->number_of_rays >>> (CUDA_rays, CUDA_triangles);

	hipMemcpy(camera->rays_through_screen, CUDA_rays, sizeof(CUDA_rays), hipMemcpyDeviceToHost);
}

int main()
{
	int a[] = { 1,2,3 };
	int b[] = { 1,2,3 };
	int c[sizeof(a) / sizeof(int)] = { 0 };

	int* cudaA = 0;
	int* cudaB = 0;
	int* cudaC = 0;

	hipMalloc(&cudaA, sizeof(a));
	hipMalloc(&cudaB, sizeof(b));
	hipMalloc(&cudaC, sizeof(c));

	hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
	hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

	vectorAdd <<< 1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);

	hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

	// Need triangle and ray
	Triangle h_triangle;
	h_triangle.vertecies[0] = { 0,0,0 };
	h_triangle.vertecies[1] = { 1,1,0 };
	h_triangle.vertecies[2] = { 2,0,0 };
	h_triangle.normal = { 0,0,-1 };

	Ray h_ray;
	h_ray.origin = { 0, 0, 1 };
	h_ray.direction = { 0,0,1 };

	Ray* d_ray;
	Triangle* d_triangle;

	hipMalloc(&d_ray, sizeof(Ray));
	hipMalloc(&d_triangle, sizeof(Triangle));

	hipMemcpy(d_ray, &h_ray, sizeof(Ray), hipMemcpyHostToDevice);
	hipMemcpy(d_triangle, &h_triangle, sizeof(Triangle), hipMemcpyHostToDevice);

	rayCastCUDA <<< 1, 1 >>> (d_ray, d_triangle);

	hipMemcpy(&h_ray, d_ray, sizeof(Ray), hipMemcpyDeviceToHost);

	std::cout << h_ray.collisionPoint.x;
	std::cout << h_ray.collisionPoint.y;
	std::cout << h_ray.collisionPoint.z;

	printf("%d\n", c[1]);

	// YOLOOO
	Camera* camera = new Camera;
	camera->scene.triangles = createTriCube(0.5f).triangles;
	camera->initialiseRaysThroughScreen();
	renderScreen(camera);

	return;

}