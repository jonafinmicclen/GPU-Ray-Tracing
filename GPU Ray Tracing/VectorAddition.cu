#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// CUDA kernel for vector addition
__global__ void addKernel(int* c, const int* a, const int* b, int size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < size) {
        c[i] = a[i] + b[i];
    }
}

// Host function to launch the CUDA kernel
hipError_t addWithCuda(int* c, const int* a, const int* b, int size) {
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Allocate GPU buffers for three vectors (a, b, and c)
    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_a);
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_a);
        hipFree(dev_b);
        return cudaStatus;
    }

    // Copy input vectors from host memory to GPU buffers
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
        return cudaStatus;
    }

    // Launch a kernel on the GPU with one thread for each element.
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    addKernel << <blocksPerGrid, threadsPerBlock >> > (dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
        return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        hipFree(dev_a);
        hipFree(dev_b);
        hipFree(dev_c);
        return cudaStatus;
    }

    // Free GPU memory
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return cudaStatus;
}